#include "hip/hip_runtime.h"
﻿#include <fstream>
#include <string>
#include <omp.h>
#include <windows.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hipfft/hipfft.h>

using namespace std;
void Check(hipError_t status)
{

	if (status != hipSuccess)
	{
		cout << "行号:" << __LINE__ << endl;
		cout << "错误:" << hipGetErrorString(status) << endl;
	}
}
template<class T>
T reverse_endian(T value)
{
	char* first = reinterpret_cast<char*>(&value);
	char* last = first + sizeof(T);
	std::reverse(first, last);
	return value;
}

bool RequireWriteData(const wchar_t* filepath, const wchar_t* flag, const wchar_t* FileMode, FILE** WriteData) {
	wchar_t DriveName[3];
	wchar_t DirectoryName[256];
	wchar_t FileName[256];
	_wsplitpath_s(filepath, DriveName, 3, DirectoryName, 256, FileName, 256, NULL, 0);
	wchar_t WritePath[260];
	wcscpy_s(WritePath, 260, DriveName);
	wcscat_s(WritePath, 260, DirectoryName);
	wcscat_s(WritePath, 260, FileName);
	wcscat_s(WritePath, 260, flag);

	errno_t error;

	if ((error = _wfopen_s(WriteData, WritePath, FileMode)) != 0) {
		return false;
	}
	return true;
}

bool TrushFile(const wchar_t* filepath, const wchar_t* flag) {
	wchar_t DriveName[3];
	wchar_t DirectoryName[256];
	wchar_t FileName[256];
	_wsplitpath_s(filepath, DriveName, 3, DirectoryName, 256, FileName, 256, NULL, 0);
	wchar_t DeletePath[260];
	wcscpy_s(DeletePath, 260, DriveName);
	wcscat_s(DeletePath, 260, DirectoryName);
	wcscat_s(DeletePath, 260, FileName);
	wcscat_s(DeletePath, 260, flag);

	if (!DeleteFile(DeletePath)) {
		return false;
	}
	return true;
}

bool WAV_Metadata(const wchar_t* filepath, unsigned __int32& samplingrate, unsigned short& bitdepth, unsigned __int32& samplesize)
{
	FILE* fprwav;
	errno_t error;
	if ((error = _wfopen_s(&fprwav, filepath, L"rb")) != 0) {
		return false;
	}

	unsigned short fmtID;
	unsigned short chnum;

	_fseeki64(fprwav, 20, SEEK_CUR);
	fread(&fmtID, 2, 1, fprwav);
	if (fmtID != 1) {
		fclose(fprwav);
		return false;
	}

	fread(&chnum, 2, 1, fprwav);
	if (chnum != 2) {
		fclose(fprwav);
		return false;
	}

	fread(&samplingrate, 4, 1, fprwav);
	if (samplingrate == 44100 || samplingrate == 44100 * 2 || samplingrate == 44100 * 4 || samplingrate == 44100 * 8 || samplingrate == 44100 * 16 ||
		samplingrate == 48000 || samplingrate == 48000 * 2 || samplingrate == 48000 * 4 || samplingrate == 48000 * 8 || samplingrate == 48000 * 16) {
	}
	else {
		fclose(fprwav);
		return false;
	}

	_fseeki64(fprwav, 6, SEEK_CUR);
	fread(&bitdepth, 2, 1, fprwav);
	if (bitdepth == 16 || bitdepth == 24 || bitdepth == 32 || bitdepth == 64) {
	}
	else {
		fclose(fprwav);
		return false;
	}
	_fseeki64(fprwav, 4, SEEK_CUR);
	fread(&samplesize, 4, 1, fprwav);

	fclose(fprwav);
	return true;
}

bool TmpWriteData(const wchar_t* filepath, unsigned short bitdepth, unsigned __int32 samplesize, FILE* tmpl, FILE* tmpr, int Times) {
	FILE* wavread;
	errno_t error;

	if ((error = _wfopen_s(&wavread, filepath, L"rb")) != 0) {
		return false;
	}

	_fseeki64(wavread, 44, SEEK_CUR);

	__int64 buffer_int = 0;
	double buffer_double = 0;
	double bit = pow(2, bitdepth - 1);
	unsigned __int64 writelength = samplesize / (bitdepth / 4);
	__int64 fillsize = 4096 * Times - writelength % (4096 * Times);
	for (int i = 0; i < fillsize; i++) {
		fwrite(&buffer_double, 8, 1, tmpl);
		fwrite(&buffer_double, 8, 1, tmpr);
	}


	for (int i = 0; i < writelength; i++) {
		fread(&buffer_int, bitdepth / 8, 1, wavread);
		buffer_int = buffer_int << (64 - bitdepth);
		buffer_int = buffer_int >> (64 - bitdepth);
		buffer_double = buffer_int / bit;
		fwrite(&buffer_double, 8, 1, tmpl);

		fread(&buffer_int, bitdepth / 8, 1, wavread);
		buffer_int = buffer_int << (64 - bitdepth);
		buffer_int = buffer_int >> (64 - bitdepth);
		buffer_double = buffer_int / bit;
		fwrite(&buffer_double, 8, 1, tmpr);
	}

	_fseeki64(tmpl, 0, SEEK_SET);
	_fseeki64(tmpr, 0, SEEK_SET);
	fclose(wavread);
	return true;
}


bool WAV_Filter_Renew(FILE* UpSampleData, FILE* OrigData, unsigned int Times, omp_lock_t* myLock) {
	omp_set_lock(myLock);

	ifstream ifs(".\\FIRFilter.dat");
	string str;
	unsigned __int64 samplesize;
	if (ifs.fail())
	{
		return false;
	}
	double* firfilter_table = new double[4095];
	unsigned __int32 i = 0;
	while (getline(ifs, str))
	{
		firfilter_table[i] = atof(str.c_str());
		i++;
	}

	ifs.close();


	ifstream ifsNS(".\\NoiseShapingCoeff.dat");
	if (ifsNS.fail())
	{
		return false;
	}
	i = 0; int s = 0;
	getline(ifsNS, str);
	unsigned int order = atoi(str.c_str());

	double** NS = new double* [2];
	NS[0] = new double[order];
	NS[1] = new double[order];
	while (getline(ifsNS, str))
	{
		if (str != "0") {
			if (i == 0)
				NS[i][s] = atof(str.c_str());
			else {
				NS[i][order - s - 1] = atof(str.c_str());
			}
			s++;
		}
		else {
			s = 0;
			i++;
		}
	}
	ifsNS.close();
	for (i = 0; i < order; i++) {
		NS[0][i] = NS[0][i] - NS[1][i];
	}

	_fseeki64(OrigData, 0, SEEK_END);
	samplesize = _ftelli64(OrigData);
	_fseeki64(OrigData, 0, SEEK_SET);
	samplesize = samplesize / 8;


	const unsigned int logtimes = unsigned int(log(Times) / log(2));
	const unsigned int fftsize = 4096 * Times;
	const unsigned int datasize = fftsize / 2;
	unsigned int* nowfftsize = new  unsigned int[logtimes];
	unsigned int* zerosize = new  unsigned int[logtimes];
	unsigned int* puddingsize = new  unsigned int[logtimes];
	unsigned int* realfftsize = new unsigned int[logtimes];
	unsigned int* addsize = new  unsigned int[logtimes];
	double** prebuffer = new double* [logtimes];
	double gain = 1;

	double* buffer = new double[fftsize];
	unsigned char* out = new unsigned char[datasize];
	for (i = 0; i < datasize; i++) {
		out[i] = 0;
	}

	double* deltabuffer = new double[order + 1];
	for (i = 0; i < order+1; i++) {
		deltabuffer[i] = 0;
	}

	double x_in = 0;
	double error_y = 0;
	double deltagain = 0.5;
	hipfftDoubleReal** fftin;
	hipfftDoubleComplex** fftout;
	hipfftDoubleReal** ifftout;
	hipfftDoubleComplex** ifftin;
	hipfftDoubleComplex** firfilter_table_fft;
	Check(hipMallocManaged((void**)&fftin, logtimes * sizeof(hipfftDoubleReal)));
	Check(hipMallocManaged((void**)&fftout, logtimes * sizeof(hipfftDoubleComplex)));
	Check(hipMallocManaged((void**)&ifftout, logtimes * sizeof(hipfftDoubleReal)));
	Check(hipMallocManaged((void**)&ifftin, logtimes * sizeof(hipfftDoubleComplex)));
	Check(hipMallocManaged((void**)&firfilter_table_fft, logtimes * sizeof(hipfftDoubleComplex)));

	hipfftHandle* FFT, * iFFT;
	Check(hipMallocManaged((void**)&FFT, logtimes * sizeof(hipfftHandle)));
	Check(hipMallocManaged((void**)&iFFT, logtimes * sizeof(hipfftHandle)));

	unsigned int p = 0;
	unsigned int k = 0;
	unsigned int t = 0;
	unsigned int q = 0;
	for (i = 1; i < Times; i = i * 2) {
		nowfftsize[p] = 4096 * (i * 2);
		realfftsize[p] = nowfftsize[p] / 2 + 1;
		zerosize[p] = nowfftsize[p] / 4;
		puddingsize[p] = realfftsize[p] - 1;
		gain = gain * (2.0 / nowfftsize[p]);
		prebuffer[p] = new double[fftsize];

		Check(hipMallocManaged((void**)&firfilter_table_fft[logtimes - p - 1], unsigned int(fftsize/i/2+1) * sizeof(hipfftDoubleComplex)));
		Check(hipMallocManaged((void**)&fftin[logtimes - p - 1], unsigned int(fftsize/ i) * sizeof(hipfftDoubleReal)));
		Check(hipMallocManaged((void**)&fftout[logtimes - p - 1], unsigned int(fftsize/i/2+1) * sizeof(hipfftDoubleComplex)));
		Check(hipMallocManaged((void**)&ifftout[logtimes - p - 1], unsigned int(fftsize / i) * sizeof(hipfftDoubleReal)));
		Check(hipMallocManaged((void**)&ifftin[logtimes - p - 1], unsigned int(fftsize/i/2+1) * sizeof(hipfftDoubleComplex)));

		for (k = 0; k < fftsize / i; k++) {
			fftin[logtimes - p - 1][k] = 0;
			ifftout[logtimes - p - 1][k] = 0;
		}
		for (k = 0; k < fftsize / i / 2 + 1; k++) {
			ifftin[logtimes - p - 1][k].x = 0;
			ifftin[logtimes - p - 1][k].y = 0;
			fftout[logtimes - p - 1][k].x = 0;
			fftout[logtimes - p - 1][k].y = 0;

		}
		for (k = 0; k < fftsize; k++) {
			prebuffer[p][k] = 0;
		}
		p++;
	}
	
	p = 0;
	for (i = 1; i < Times; i = i * 2) {
		hipfftPlan1d(&FFT[logtimes - p - 1], fftsize/i, HIPFFT_D2Z, 1);
		hipDeviceSynchronize();
		hipfftPlan1d(&iFFT[logtimes - p - 1], fftsize/i, HIPFFT_Z2D, 1);
		hipDeviceSynchronize();
		p++;
	}

	for (k = 0; k < logtimes; k++) {
		for (i = 0; i < 4095; i++) {
			fftin[k][i] = firfilter_table[i];
		}
		for (i = 4095; i < nowfftsize[k]; i++) {
			fftin[k][i] = 0;
		}
	}

	for (k = 0;k < logtimes;k++) {
		hipfftExecD2Z(FFT[logtimes - k - 1], fftin[logtimes - k - 1], fftout[logtimes - k - 1]);
		hipDeviceSynchronize();
		for (p = 0; p < realfftsize[logtimes - k - 1]; p++) {
			firfilter_table_fft[logtimes - k - 1][p].x = fftout[logtimes - k - 1][p].x;
			firfilter_table_fft[logtimes - k - 1][p].y = fftout[logtimes - k - 1][p].y;
		}
	}

	unsigned __int64 SplitNum = unsigned __int64((samplesize / datasize) * Times);
	deltagain = gain * deltagain;
	omp_unset_lock(myLock);
	for (k = 0; k < SplitNum; k++) {
		fread(buffer, 8, datasize / Times, OrigData);
		for (t = 0;t < logtimes;t++) {
			q = 0;
			for (p = 0; p < zerosize[t]; p++) {
				fftin[t][q] = buffer[p];
				q++;
				fftin[t][q] = 0;
				q++;
			}
			memset(fftin[t] + q, 0, 8 * (nowfftsize[t] - q));
			hipfftExecD2Z(FFT[t], fftin[t], fftout[t]);
			hipDeviceSynchronize();
			for (p = 0; p < realfftsize[t]; p++) {
				ifftin[t][p].x = fftout[t][p].x * firfilter_table_fft[t][p].x - fftout[t][p].y * firfilter_table_fft[t][p].y;
				ifftin[t][p].y = fftout[t][p].x * firfilter_table_fft[t][p].y + firfilter_table_fft[t][p].x * fftout[t][p].y;
			}
			hipfftExecZ2D(iFFT[t], ifftin[t], ifftout[t]);
			hipDeviceSynchronize();
			for (p = 0; p < realfftsize[t]; p++) {
				ifftout[t][p] = ifftout[t][p] / (fftsize / i);
			}
			for (p = 0; p < puddingsize[t]; p++) {
				buffer[p] = prebuffer[t][p] + ifftout[t][p];
			}
			q = 0;
			for (p = puddingsize[t]; p < nowfftsize[t]; p++) {
				prebuffer[t][q] = ifftout[t][p];
				q++;
			}
		}

		for (q = 0; q < datasize; q++) {

			x_in = buffer[q] * deltagain;

			for (t = 0; t < order; t++) {
				x_in += NS[0][t] * deltabuffer[t];
			}

			if (x_in >= 0.0) {
				out[q] = 1;
				error_y = -1.0;
			}
			else {
				out[q] = 0;
				error_y = 1.0;
			}
			for (t = order; t > 0; t--) {
				deltabuffer[t] = deltabuffer[t - 1];
			}

			deltabuffer[0] = x_in + error_y;

			for (t = 0; t < order; t++) {
				deltabuffer[0] += NS[1][t] * deltabuffer[t + 1];
			}
		}
		fwrite(out, 1, datasize, UpSampleData);
	}


	for (i = 0; i < logtimes; i++) {
		delete[] prebuffer[i];
	}

	delete[] NS[0];
	delete[] NS[1];
	delete[] NS;
	delete[] nowfftsize;
	delete[] zerosize;
	delete[] puddingsize;
	delete[] realfftsize;
	delete[] out;
	delete[] prebuffer;
	delete[] buffer;
	delete[] deltabuffer;
	delete[] firfilter_table;
	return true;
}

bool DSD_Write(FILE* LData, FILE* RData, FILE* WriteData, unsigned int DSDSamplingRate, unsigned short bitdepth, unsigned __int32 samplesize, unsigned int Times) {
	unsigned __int64 writelength = samplesize / (bitdepth / 4);
	unsigned __int64 DSD_SampleSize = writelength * Times;
	unsigned __int64 DSD_DataSize = DSD_SampleSize / 4;
	_fseeki64(LData, 0, SEEK_END);
	_fseeki64(RData, 0, SEEK_END);
	_fseeki64(LData, _ftelli64(LData) - DSD_SampleSize, SEEK_SET);
	_fseeki64(RData, _ftelli64(RData) - DSD_SampleSize, SEEK_SET);

	fwrite("FRM8", 4, 1, WriteData);
	unsigned __int64 binary = 0;
	unsigned short ushort = 0;
	unsigned char uchar = 0;
	unsigned __int64 ulong = 0;
	binary = reverse_endian(DSD_DataSize + 152);
	fwrite(&binary, 8, 1, WriteData);
	fwrite("DSD ", 4, 1, WriteData);
	fwrite("FVER", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(4);
	fwrite(&binary, 4, 1, WriteData);


	binary = 1;
	fwrite(&binary, 1, 1, WriteData);
	binary = 5;
	fwrite(&binary, 1, 1, WriteData);
	binary = 0;
	fwrite(&binary, 1, 1, WriteData);
	binary = 0;
	fwrite(&binary, 1, 1, WriteData);

	fwrite("PROP", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(108);
	fwrite(&binary, 4, 1, WriteData);
	fwrite("SND ", 4, 1, WriteData);

	fwrite("FS  ", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(4);
	fwrite(&binary, 4, 1, WriteData);
	unsigned __int32 binary1;
	binary1 = reverse_endian(DSDSamplingRate);
	fwrite(&binary1, 4, 1, WriteData);

	fwrite("CHNL", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(10);
	fwrite(&binary, 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 1, 1, WriteData);
	binary = 2;
	fwrite(&binary, 1, 1, WriteData);
	fwrite("SLFT", 4, 1, WriteData);
	fwrite("SRGT", 4, 1, WriteData);

	fwrite("CMPR", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(20);
	fwrite(&binary, 4, 1, WriteData);

	fwrite("DSD ", 4, 1, WriteData);
	binary = 14;
	fwrite(&binary, 1, 1, WriteData);
	fwrite("not compressed ", 15, 1, WriteData);

	fwrite("ABSS", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(8);
	fwrite(&binary, 4, 1, WriteData);
	fwrite(&ushort, 2, 1, WriteData);
	fwrite(&uchar, 1, 1, WriteData);
	fwrite(&uchar, 1, 1, WriteData);
	fwrite(&ulong, 4, 1, WriteData);

	fwrite("LSCO", 4, 1, WriteData);
	binary = 0;
	fwrite(&binary, 4, 1, WriteData);
	binary = reverse_endian(2);
	fwrite(&binary, 4, 1, WriteData);
	fwrite(&ushort, 2, 1, WriteData);

	fwrite("DSD ", 4, 1, WriteData);
	binary = reverse_endian(DSD_DataSize);
	fwrite(&binary, 8, 1, WriteData);
	unsigned __int64 i = 0;
	unsigned char* onebyte = new unsigned char[2];
	unsigned char* tmpdataL = new unsigned char[8];
	unsigned char* tmpdataR = new unsigned char[8];

	for (i = 0; i < DSD_SampleSize / 8; i++) {
		fread(tmpdataL, 1, 8, LData);
		fread(tmpdataR, 1, 8, RData);
		onebyte[0] = tmpdataL[0] << 7;
		onebyte[0] += tmpdataL[1] << 6;
		onebyte[0] += tmpdataL[2] << 5;
		onebyte[0] += tmpdataL[3] << 4;
		onebyte[0] += tmpdataL[4] << 3;
		onebyte[0] += tmpdataL[5] << 2;
		onebyte[0] += tmpdataL[6] << 1;
		onebyte[0] += tmpdataL[7] << 0;
		onebyte[1] = tmpdataR[0] << 7;
		onebyte[1] += tmpdataR[1] << 6;
		onebyte[1] += tmpdataR[2] << 5;
		onebyte[1] += tmpdataR[3] << 4;
		onebyte[1] += tmpdataR[4] << 3;
		onebyte[1] += tmpdataR[5] << 2;
		onebyte[1] += tmpdataR[6] << 1;
		onebyte[1] += tmpdataR[7] << 0;
		fwrite(onebyte, 1, 2, WriteData);
	}
	delete[] onebyte;
	delete[] tmpdataL;
	delete[] tmpdataR;
	return true;
}

bool WAV_Convert(const wchar_t* filepath, unsigned int DSD_Times) {
	unsigned __int32 samplingrate;
	unsigned short bitdepth;
	unsigned __int32 samplesize;
	if (!WAV_Metadata(filepath, samplingrate, bitdepth, samplesize))
		return false;
	unsigned int Times;
	unsigned int DSDSamplingRate;
	if (0 == samplingrate % 44100) {
		Times = DSD_Times / (samplingrate / 44100);
		DSDSamplingRate = samplingrate * Times;
	}
	else {
		Times = DSD_Times / (samplingrate / 48000);
		DSDSamplingRate = samplingrate * Times;
	}
	bool flag = true;
	bool flagl = true;
	bool flagr = true;
	FILE* tmpl;
	FILE* tmpr;

	if (!RequireWriteData(filepath, L"_tmpL0", L"wb", &tmpl)) {
		flagl = false;
		flag = false;
	}
	if (!RequireWriteData(filepath, L"_tmpR0", L"wb", &tmpr)) {
		flagr = false;
		flag = false;
	}
	if (flag)if (!TmpWriteData(filepath, bitdepth, samplesize, tmpl, tmpr, Times)) {
		flag = false;
	}
	if (flagl) {
		fclose(tmpl);
	}
	if (flagr) {
		fclose(tmpr);
	}
	omp_lock_t myLock;
	omp_init_lock(&myLock);
#pragma omp parallel
#pragma omp sections
	{
#pragma omp section
		{

			if (flag) {
				bool flagUpl = true;
				bool flagOrigl = true;
				FILE* tmpl;
				FILE* UpsampleDataL;
				omp_set_lock(&myLock);
				if (!RequireWriteData(filepath, L"_tmpL0", L"rb", &tmpl)) {
					flagOrigl = false;
					flag = false;
				}
				if (!RequireWriteData(filepath, L"_tmpLDSD", L"wb", &UpsampleDataL)) {
					flagUpl = false;
					flag = false;
				}
				omp_unset_lock(&myLock);
				if (flag)if (!WAV_Filter_Renew(UpsampleDataL, tmpl, Times, &myLock)) {
					flag = false;
				}
				if (flagUpl) {
					fclose(UpsampleDataL);
				}
				if (flagOrigl) {
					fclose(tmpl);
				}
			}
		}
#pragma omp section  
		{

			if (flag) {
				bool flagUpr = true;
				bool flagOrigr = true;
				FILE* tmpr;
				FILE* UpsampleDataR;
				omp_set_lock(&myLock);
				if (!RequireWriteData(filepath, L"_tmpR0", L"rb", &tmpr)) {
					flagOrigr = false;
					flag = false;
				}
				if (!RequireWriteData(filepath, L"_tmpRDSD", L"wb", &UpsampleDataR)) {
					flagUpr = false;
					flag = false;
				}
				omp_unset_lock(&myLock);
				if (flag)if (!WAV_Filter_Renew(UpsampleDataR, tmpr, Times, &myLock)) {
					flag = false;
				}
				if (flagUpr) {
					fclose(UpsampleDataR);
				}
				if (flagOrigr) {
					fclose(tmpr);
				}
			}
		}
	}
	omp_destroy_lock(&myLock);

	if (flag) {
		FILE* tmpDSD;
		bool flagOrigl = true;
		bool flagOrigr = true;
		bool flagDSD = true;
		if (!RequireWriteData(filepath, L"_tmpLDSD", L"rb", &tmpl)) {
			TrushFile(filepath, L"_tmpLDSD");
			flagOrigl = false;
			flag = false;
		}
		if (!RequireWriteData(filepath, L"_tmpRDSD", L"rb", &tmpr)) {
			TrushFile(filepath, L"_tmpRDSD");
			flagOrigr = false;
			flag = false;
		}
		if (!RequireWriteData(filepath, L".dff", L"wb", &tmpDSD)) {
			TrushFile(filepath, L"_tmpLDSD");
			TrushFile(filepath, L"_tmpRDSD");
			flagDSD = false;
			flag = false;
		}
		if (flag)if (!DSD_Write(tmpl, tmpr, tmpDSD, DSDSamplingRate, bitdepth, samplesize, Times)) {
			TrushFile(filepath, L"_tmpLDSD");
			TrushFile(filepath, L"_tmpRDSD");
		}
		if (flagOrigl) {
			fclose(tmpl);
		}
		if (flagOrigr) {
			fclose(tmpr);
		}
		if (flagDSD) {
			fclose(tmpDSD);
		}
	}

	TrushFile(filepath, L"_tmpL0");
	TrushFile(filepath, L"_tmpR0");
	TrushFile(filepath, L"_tmpLDSD");
	TrushFile(filepath, L"_tmpRDSD");

	if (!flag) {
		return false;
	}
	return true;
}

int wmain(int argc, wchar_t* argv[])
{
	if (argc != 3)
		return -1;
	unsigned int DSD_Times = _wtoi(argv[2]);
	wchar_t cmdflac[260] = LR"(flac -d -f ")";
	wcscat_s(cmdflac, 260, argv[1]);
	wcscat_s(cmdflac, 260, LR"(")");
	STARTUPINFO si;
	PROCESS_INFORMATION pi;
	ZeroMemory(&si, sizeof(si));
	ZeroMemory(&pi, sizeof(pi));
	DWORD flacExitCode;
	DWORD wvExitCode;

	if (CreateProcess(NULL, (LPWSTR)cmdflac, NULL, NULL, false, 0, NULL, NULL, &si, &pi))
	{
		CloseHandle(pi.hThread);
		WaitForSingleObject(pi.hProcess, INFINITE);
		GetExitCodeProcess(pi.hProcess, &flacExitCode);
		CloseHandle(pi.hProcess);
	}
	ZeroMemory(&si, sizeof(si));
	ZeroMemory(&pi, sizeof(pi));
	if (!flacExitCode)
	{
		wchar_t DriveName[3];
		wchar_t DirectoryName[256];
		wchar_t FileName[256];
		_wsplitpath_s(argv[1], DriveName, 3, DirectoryName, 256, FileName, 256, NULL, 0);
		wchar_t ReadPath[260];
		wcscpy_s(ReadPath, 260, DriveName);
		wcscat_s(ReadPath, 260, DirectoryName);
		wcscat_s(ReadPath, 260, FileName);
		wchar_t wavPath[260];
		wcscpy_s(wavPath, 260, ReadPath);
		wcscat_s(wavPath, 260, LR"(.wav)");
		if (WAV_Convert(wavPath, DSD_Times)) {
			DeleteFile(wavPath);
			wchar_t cmdwv[260] = LR"(wavpack -d -h ")";
			wcscat_s(cmdwv, 260, ReadPath);
			wcscat_s(cmdwv, 260, LR"(.dff")");
			if (CreateProcess(NULL, (LPWSTR)cmdwv, NULL, NULL, false, 0, NULL, NULL, &si, &pi))
			{
				CloseHandle(pi.hThread);
				WaitForSingleObject(pi.hProcess, INFINITE);
				GetExitCodeProcess(pi.hProcess, &wvExitCode);
				CloseHandle(pi.hProcess);
			}
			ZeroMemory(&si, sizeof(si));
			ZeroMemory(&pi, sizeof(pi));
			if (!wvExitCode)
				return 0;
		}
	}
}
